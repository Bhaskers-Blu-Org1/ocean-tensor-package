/* ------------------------------------------------------------------------ */
/* Copyright 2018, IBM Corp.                                                */
/*                                                                          */
/* Licensed under the Apache License, Version 2.0 (the "License");          */
/* you may not use this file except in compliance with the License.         */
/* You may obtain a copy of the License at                                  */
/*                                                                          */
/*    http://www.apache.org/licenses/LICENSE-2.0                            */
/*                                                                          */
/* Unless required by applicable law or agreed to in writing, software      */
/* distributed under the License is distributed on an "AS IS" BASIS,        */
/* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. */
/* See the License for the specific language governing permissions and      */
/* limitations under the License.                                           */
/* ------------------------------------------------------------------------ */

#ifndef SD_TEMPLATE_FILE
#define SD_TEMPLATE_FILE "core/gpu/template_index.cu"

#include "solid/base/generic/dtype_macros.h"
#include "solid/base/gpu/dtype_gpu.h"
#include "solid/core/gpu/apply_elemwise1.h"
#include "solid/core/gpu/apply_elemwise2.h"
#include "solid/core/gpu/index1.h"
#include "solid/core/gpu/index2.h"

#include "solid/base/generic/generate_all_types.h"
#else


/* ============================================================================== */
/* Function definition - Add if negative                                          */
/* ============================================================================== */

#if (SDTYPE_IS_SIGNED_INT(SDXTYPE))
/* Create the cuda kernels */
SOLID_KERNELS_ELEMWISE1_PARAM(UNROLL, addIfNegative,
                              { SOLID_C_TYPE(SDXTYPE) value; },
                              { if (*_ptr < 0) *_ptr += param.value; })

/* ------------------------------------------------------------------------ */
SOLID_API int SOLID_FUNCTION(add_if_negative)(int ndims, const size_t *size,
                                              const ptrdiff_t *strides, void *ptr,
                                              const solid_scalar scalar,
                                              hipStream_t stream)
/* ----------------------------------------------------------------------- */
{  SOLID_KERNEL_PARAM(addIfNegative) param;
   int result = 0;

   /* Set user parameters*/
   param.value = SOLID_SCALAR_C_VALUE(scalar);

   /* Set up and launch the appropriate kernel */
   SOLID_LAUNCH_ELEMWISE1_PARAM(UNROLL, addIfNegative, 0, stream, param, result);

   return result;
}
#endif




/* ============================================================================== */
/* Function definition - Index to offset                                          */
/* ============================================================================== */

#if (SDTYPE_IS_INT(SDXTYPE))
/* Create the cuda kernels */
SOLID_KERNELS_ELEMWISE2_FULL(SDXTYPE, int64, 1, NO_UNROLLING, indexToOffset, \
                              { solid_int64 strides[SOLID_MAX_TENSOR_DIMS]; \
                                ptrdiff_t   strideReduce; \
                                int         nstrides; \
                              }, \
                              {  solid_int64 s = 0; \
                                 int i; \
                                 for (i = 0; i < param.nstrides; i++) \
                                 {  s += param.strides[i] * *((SOLID_C_TYPE(SDXTYPE) *)(((char *)_ptr1) + i * param.strideReduce)); \
                                 } \
                                 *_ptr2 = s; \
                              })


/* ------------------------------------------------------------------------ */
SOLID_API int SOLID_FUNCTION(index_to_offset)(int nstrides, solid_int64 *strides,
                                              ptrdiff_t strideReduce, size_t nelem,
                                              ptrdiff_t stride1, void *ptr1,
                                              ptrdiff_t stride2, void *ptr2,
                                              hipStream_t stream)
/* ----------------------------------------------------------------------- */
{  SOLID_KERNEL_PARAM(indexToOffset) param;
   ptrdiff_t *strides1 = &stride1;
   ptrdiff_t *strides2 = &stride2;
   size_t    *size     = &nelem;
   int        ndims    = 1;
   int        result   = 0;
   int        i;

   /* Set user parameters*/
   for (i = 0; i < nstrides; i++) param.strides[i] = strides[i];
   param.strideReduce = strideReduce;
   param.nstrides     = nstrides;

   /* Set up and launch the appropriate kernel */
   SOLID_LAUNCH_ELEMWISE2_FULL(SDXTYPE, int64, 1, NO_UNROLLING, indexToOffset, 0, stream, param, result);

   return result;
}
#endif





/* ============================================================================== */
/* Function definitions - Get index                                               */
/* ============================================================================== */

/* Create the cuda kernel */
SOLID_KERNELS_INDEX2(getIndex, { *_ptr2 = *_ptr1; })

/* -------------------------------------------------------------------- */
SOLID_API int SOLID_FUNCTION(get_index)(int ndims, const size_t *size, solid_int64 **offsets,
                                        const ptrdiff_t *strides1, void *ptr1,
                                        const ptrdiff_t *strides2, void *ptr2,
                                        hipStream_t stream)
/* -------------------------------------------------------------------- */
{  int result = 0;

   /* Set up and launch the appropriate kernel */
   SOLID_LAUNCH_INDEX2(getIndex, stream, result);

   return result;
}


/* ============================================================================== */
/* Function definitions - Set index                                               */
/* ============================================================================== */

/* Create the cuda kernel */
SOLID_KERNELS_INDEX2(setIndex, { *_ptr1 = *_ptr2; })

/* -------------------------------------------------------------------- */
SOLID_API int SOLID_FUNCTION(set_index)(int ndims, const size_t *size, solid_int64 **offsets,
                                        const ptrdiff_t *strides1, void *ptr1,
                                        const ptrdiff_t *strides2, void *ptr2,
                                        hipStream_t stream)
/* -------------------------------------------------------------------- */
{  int result = 0;

   /* Set up and launch the appropriate kernel */
   SOLID_LAUNCH_INDEX2(setIndex, stream, result);

   return result;
}


/* ============================================================================== */
/* Function definitions - Fill index                                              */
/* ============================================================================== */

/* Create the cuda kernels */
SOLID_KERNELS_INDEX1_PARAM(fill_index,
                           { SOLID_C_TYPE(SDXTYPE) value; },
                           { *_ptr = param.value; })

/* ------------------------------------------------------------------------ */
SOLID_API int SOLID_FUNCTION(fill_index)(int ndims, const size_t *size, solid_int64 **offsets,
                                         const ptrdiff_t *strides, void *ptr,
                                         const solid_scalar scalar, hipStream_t stream)
/* ----------------------------------------------------------------------- */
{  SOLID_KERNEL_PARAM(fill_index) param;
   int result = 0;

   /* Set user parameters*/
   param.value = SOLID_SCALAR_C_VALUE(scalar);

   /* Set up and launch the appropriate kernel */
   SOLID_LAUNCH_INDEX1_PARAM(fill_index, stream, param, result);

   return result;
}


#endif
